#include <iostream>
#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>


/**
 * This kernel essentially serves as a "srand(seed)" on the GPU
 */
__global__ void setup_kernel(hiprandState * state){
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
        int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int id = x + (blockDim.x * gridDim.x * y);

	hiprand_init(420+69, id, 0, &state[id]);
}

/**
 * Executes one timestep on a thread
 */
__global__ void update(short * inGrid, short * outGrid, hiprandState * rand_state){
	short y = (blockIdx.y * blockDim.y) + threadIdx.y;
	short x = (blockIdx.x * blockDim.x) + threadIdx.x;
	short N = blockDim.x * gridDim.x;
	short id = x + (N * y);

	//generate random number
	unsigned int randres = hiprand(&rand_state[id]);
	randres = (short) (((int) randres) % 300);

	short index = 0;
	short state = inGrid[id];
	
	//calculate index
	index += inGrid[((x-1) % N) + N*y];
	index += inGrid[((x+1) % N) + N*y];
	index += inGrid[x + N*((y-1) % N)];
	index += inGrid[x - N*((y+1) % N)];
	index += inGrid[((x+1) % N) + N*((y-1) % N)];
	index += inGrid[((x-1) % N) + N*((y-1) % N)];
	index += inGrid[((x+1) % N) + N*((y+1) % N)];
	index += inGrid[((x-1) % N) + N*((y+1) % N)];
	
	//find new state
	if(state == 0){
		if(randres == 0) state = 2;
		else if(index < 7) state = 0;	
		else if(index < 17) state = 1;
		else state = 3;
	}
	else if(state == 1){
		if(randres == 0 || index > 16) state = 3;
		else if(index < 1) state = 0;
		else state = 1;
	}
	else if(state == 2){
		if(randres % 5 < 2) state = 2;
		else if(randres % 5 < 4) state = 1;
		else state = 0;
	}
	else if(state == 3){
		if(index > 9) state = 3;
		else state = 1;
	}
	
	//update relevant array
	outGrid[id] = state;
}


void print_grid(short * grid, short* source_d, int N) {
	hipMemcpy(grid, source_d, N*N*sizeof(short), hipMemcpyDeviceToHost);
	
	for(int y = 0; y < N; y++){
		for(int x = 0; x < N; x++){
			printf("%d ", grid[x + N*y]);
		}
		printf("\n");
	}
	
	return;
}

int main(int argc, char * argv[]){
	
	if(argc < 4){
		printf("usage: ./simCuda <sidelength> <timesteps> <block divisor for each dimention>\n");
		exit(1);
	}
	
	//arguments(board and GPU dimensions)
	const int N = atoi(argv[1]);
	const int t = atoi(argv[2]);	
	const int common_divisor = atoi(argv[3]);

	if(N*N % 32){
		printf("Choose multiple of 32 on sides for best results\n");
	}
	
	if(N % common_divisor){
		printf("Try again with divisor of sidelength\n");
		exit(1);
	} 	
	
	printf("begin grid setup\n");	
	//*******************************************************************
	//****************************GRID SETUP*****************************
	//*******************************************************************
	int blockw = N/common_divisor;

	//blank grid (host)
	short * blankGrid;

	//grid memory allocation (host)
	blankGrid = (short*) calloc(N*N, sizeof(short));
	printf("grid host allocation complete\n");

	//grids (device)
	short * evenGrid_d;
	short * oddGrid_d;

	//memory allocation (device)
	hipMalloc((void**) &evenGrid_d, N*N*sizeof(short));
	hipMalloc((void**) &oddGrid_d, N*N*sizeof(short));

	printf("even/odd grid device allocation complete\n");
	
	//transfer CPU contents to GPU (all zeroed out)
	for(int y = 0; y < N; y++){
		for(int x = 0; x < N; x++){
			hipMemcpy(evenGrid_d, blankGrid, N*N*sizeof(short), hipMemcpyHostToDevice);
			hipMemcpy(oddGrid_d, blankGrid, N*N*sizeof(short), hipMemcpyHostToDevice);
		}
	}
	
	printf("device grid initialization complete\n");


	//*******************************************************************
	//***************************KERNEL CALLS****************************
	//*******************************************************************
		
	printf("begin kernel calls\n");
	//declare dimentions of blocks and block arrangement
	dim3 BLOCK_ARRANGEMENT(common_divisor,common_divisor,1);
	dim3 BLOCK_SHAPE(blockw,blockw,1);

	//random number stuff (it's a headache)
	
	hiprandState * states_d;
	hipMalloc((void**) &states_d, N*N*sizeof(hiprandState));
	printf("device random memory allocation complete\n");

	setup_kernel<<<BLOCK_ARRANGEMENT,BLOCK_SHAPE>>>(states_d);
	printf("device random initialization complete, begin sim\n");	

	//call updates for each timestep
	for(int i = 1; i < t+1; i++){
		if(i%2==0){
			update<<<BLOCK_ARRANGEMENT,BLOCK_SHAPE>>>(evenGrid_d, oddGrid_d, states_d);
			print_grid(blankGrid, oddGrid_d, N);
			for(int i = 0; i < N; i++) printf("*");
			printf("\n");
		}
		else{
			update<<<BLOCK_ARRANGEMENT,BLOCK_SHAPE>>>(oddGrid_d, evenGrid_d, states_d);
			print_grid(blankGrid, evenGrid_d, N);
			for(int i = 0; i < N; i++) printf("*");
			printf("\n");
		}
		//usleep(300000);
	}

	//*******************************************************************
	//*****************************CLEANUP*******************************
	//*******************************************************************
	
	hipFree(states_d);
        hipFree(evenGrid_d);
	hipFree(oddGrid_d);
	free(blankGrid);

	return 0;
}
